#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>


extern "C" { 
#include "cpu_functions.h"
}
#include "conv_kernels.cuh"

#define N 2048  
#define K 5  
#define M 2048 
#define BLOCK_SIZE 16

__constant__ float window_cm[K*K];

int main() {
    int size_A = M * N * sizeof(float);
    int size_B = K * K * sizeof(float);
    int size_C = (M + 1 - K) * (N + 1 - K) * sizeof(float);

    // Allocate host memory (for cpu benchmarks)
    float *h_A = (float*)malloc(size_A);
    float *h_B = (float*)malloc(size_B);
    float *h_C = (float*)malloc(size_C);
    float *h_window = (float*)malloc(size_B);

    // Initialize matrices
    initMatrix(h_A, M, K);
    initMatrix(h_B, K, K);
    initMatrix(h_window, K, K);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(window_cm), h_window, size_B);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsed_time;
    float repeats = 100.0f;
    long long flops = 2LL * (M + 1 - K) * (N + 1 - K) * K * K;

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N - K + BLOCK_SIZE) / BLOCK_SIZE, (M - K + BLOCK_SIZE) / BLOCK_SIZE);
    dim3 blockDim1D(BLOCK_SIZE * BLOCK_SIZE);
    
    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        conv_naive<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        conv_gmc<<<gridDim, blockDim1D>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        conv_cm<<<gridDim, blockDim>>>(d_A, d_C, N, K, M);
        hipDeviceSynchronize();
        conv_shared<<<gridDim, blockDim>>>(d_A, d_C, N, K, M);
        hipDeviceSynchronize();
    }

    // Implementation 1
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        conv_naive<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(naive) Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time
    );

    // Implementation 2
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        conv_gmc<<<gridDim, blockDim1D>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(gmc) Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time
    );

    // Implementation 3
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        conv_cm<<<gridDim, blockDim>>>(d_A, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(cm) Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time
    );

    // Implementation 4
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        conv_cm<<<gridDim, blockDim>>>(d_A, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(shared) Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time
    );
    
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }


    // Free memory
    free(h_A);
    free(h_B);
    free(h_window);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(window_cm);
}


