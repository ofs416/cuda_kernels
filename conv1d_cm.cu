#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

extern "C" { 
#include "cpu_functions.h"
}

#define BLOCK_SIZE 16

// Constant memory
__constant__ float filter_cm[5];

__global__ void conv_1dhz_cm(float *input, float *output, int width,
                                                     int height, int f_size) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        float sum = 0.0f;
        for (int j = 0; j < f_size; j++) {
            int input_col = col + j - f_size / 2;
            if (input_col >= 0 && input_col < width) {
                sum += input[row * width + input_col] * filter_cm[j];
            }
            // Implicit zero-padding: we don't add anything for out-of-bounds inputs
        }
        output[width * row + col] = sum;
    }
}


void check_cuda_error(hipError_t error, const char *function_name) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function_name, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Image and kernel parameters
    const unsigned int width = 4096;
    const unsigned int height = 4096;
    const unsigned int filter_size = 5;
    const unsigned int image_size = width * height * sizeof(float);

    // Allocate host memory
    float *h_input = (float*)malloc(image_size);
    float *h_output = (float*)malloc(image_size);
    float *h_output_gpu = (float*)malloc(image_size);
    float *h_output_cpu = (float*)malloc(image_size);
    float *h_filter = (float*)malloc(filter_size * sizeof(float));

    // Initialize input image and kernel (example initialization)
    initMatrix(h_input, height, width);
    initMatrix(h_filter, filter_size, 1);

    // Allocate device memory
    float *d_input, *d_output;
    check_cuda_error(hipMalloc(&d_input, image_size), "hipMalloc d_input");
    check_cuda_error(hipMalloc(&d_output, image_size), "hipMalloc d_output");

    // Copy input data to device
    check_cuda_error(hipMemcpy(d_input, h_input, image_size, hipMemcpyHostToDevice), "hipMemcpy H2D input");
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(filter_cm), h_filter, filter_size * sizeof(float)), "hipMemcpyToSymbol kernel");

    // Launch kernel
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Check computation
    // Compute convolution on CPU
    conv_1dhz_cpu(h_input, h_output_cpu, width, height, h_filter, filter_size);
    // compute convolution with custom kernel
    conv_1dhz_cm<<<gridDim, blockDim>>>(d_input, d_output, width, height, filter_size);
    // Check for kernel launch errors
    check_cuda_error(hipGetLastError(), "Kernel launch");
    // Wait for kernel to finish
    check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    // Copy result back to host
    check_cuda_error(hipMemcpy(h_output_gpu, d_output, image_size, hipMemcpyDeviceToHost), "hipMemcpy D2H output");
    // Compare CPU and GPU results
    if (compare_results(h_output_cpu, h_output_gpu, width, height, 1e-5f)) {
        printf("CPU and GPU results match!\n");
    } else {
        printf("CPU and GPU results do not match.\n");
    }

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        conv_1dhz_cm<<<gridDim, blockDim>>>(d_input, d_output, width, height, filter_size);
        // Wait for kernel to finish
        check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    }
    

    // Create events
    hipEvent_t start, stop;
    check_cuda_error(hipEventCreate(&start), "create event start");
    check_cuda_error(hipEventCreate(&stop), "create event stop");
    // Benchmark
    float elapsed_time;
    float repeats = 100.0f;
    long long flops = 2LL * width * height * filter_size;
    check_cuda_error(hipEventRecord(start), "start event recording");
    for (int i = 0; i < repeats; i++) {
        conv_1dhz_cm<<<gridDim, blockDim>>>(d_input, d_output, width, height, filter_size);
    }
    check_cuda_error(hipEventRecord(stop), "stop event recording");
    check_cuda_error(hipEventSynchronize(start), "hipDeviceSynchronize");
    check_cuda_error(hipEventSynchronize(stop), "hipDeviceSynchronize");
    check_cuda_error(hipEventElapsedTime(&elapsed_time, start, stop), "elapsed time");
    printf(
        "Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time
    );
    

    // Free memory
    check_cuda_error(hipFree(d_input), "hipFree d_input");
    check_cuda_error(hipFree(d_output), "hipFree d_output");
    check_cuda_error(hipEventDestroy(start), "hipEventDestroy start");
    check_cuda_error(hipEventDestroy(stop), "hipEventDestroy stop");
    free(h_input);
    free(h_output);
    free(h_filter);
  

    return 0;
}