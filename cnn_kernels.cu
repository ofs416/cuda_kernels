#include "hip/hip_runtime.h"
// Implementation of CNN kernel following based on the optimisations seen in gpu_functions.cu

#include <hip/hip_runtime.h>
#include "cnn_kernels.cuh"

#define BLOCK_SIZE 32

// Naive CUDA kernel for CNN 
// Kernel (matrix B) size of k (heed attention to context of the work kernel)
// Input matrix B with size m X n
// Output size of m + 1 - k x n + 1 - k
__global__ void cnn (float *A, float *B, float *C, uint n, uint k, uint m) {
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = - (int)(k / 2); i < (int)(k / 2); i++) {
            for (int j = - (int)(k / 2) ; j < (int)(k / 2); j++) {
                 sum += A[k * i + j] * B[n * (row + i) + col + j];
            }
        }
        C[n * row + col] = sum;
    }
}