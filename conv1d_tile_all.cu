#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

extern "C" { 
#include "cpu_functions.h"
}

#define BLOCK_SIZE 1024
#define MAX_FILTER_SIZE 63
#define ELEMENTS_PER_THREAD 2
#define ACTUAL_BLOCK_SIZE (BLOCK_SIZE/ELEMENTS_PER_THREAD) // Now 64 threads per block
#define SMEM_SIZE (BLOCK_SIZE + MAX_FILTER_SIZE - 1)

__constant__ float filter_cm[MAX_FILTER_SIZE];

__global__ void conv_1d_tile_all(float *input, float *output, int width, 
                                        int height, int filter_size, bool transpose) {
    extern __shared__ float shared_mem[];
    
    const int tid = threadIdx.x;
    const int row = blockIdx.y;
    // Each thread now starts at a position that's ELEMENTS_PER_THREAD times further apart
    const int base_col = blockIdx.x * BLOCK_SIZE + tid * ELEMENTS_PER_THREAD;
    const int radius = filter_size / 2;
    
    int trans_width = transpose ? height : width;
    int trans_height = transpose ? width : height;
    int input_row = transpose ? base_col : row;
    
    if (input_row >= trans_height) return;

    // Load data into shared memory
    const int block_start = blockIdx.x * BLOCK_SIZE - radius;
    
    // Each thread now loads ELEMENTS_PER_THREAD elements at a time
    // We need to load BLOCK_SIZE + filter_size - 1 total elements
    const int elements_to_load = BLOCK_SIZE + filter_size - 1;
    const int loads_per_thread = (elements_to_load + ACTUAL_BLOCK_SIZE - 1) / ACTUAL_BLOCK_SIZE;
    
    #pragma unroll
    for (int i = 0; i < loads_per_thread; i++) {
        int load_idx = tid + (i * ACTUAL_BLOCK_SIZE);
        if (load_idx < elements_to_load) {
            int global_idx = block_start + load_idx;
            if (global_idx >= 0 && global_idx < trans_width) {
                shared_mem[load_idx] = input[input_row * trans_width + global_idx];
            } else {
                shared_mem[load_idx] = 0.0f;
            }
        }
    }
    __syncthreads();

    // Each thread processes ELEMENTS_PER_THREAD consecutive elements
    #pragma unroll
    for (int offset = 0; offset < ELEMENTS_PER_THREAD; offset++) {
        int input_col = base_col + offset;
        
        if (input_col < trans_width) {
            float sum = 0.0f;
            
            // Compute convolution for this element
            #pragma unroll
            for (int i = 0; i < filter_size; i++) {
                sum += shared_mem[tid * ELEMENTS_PER_THREAD + offset + i] * filter_cm[i];
            }
            
            // Store result
            if (transpose) {
                output[input_col * trans_height + input_row] = sum;
            } else {
                output[input_row * trans_width + input_col] = sum;
            }
        }
    }
}



void check_cuda_error(hipError_t error, const char *function_name) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function_name, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Image and kernel parameters
    const unsigned int width = 4096;
    const unsigned int height = 4096;
    const unsigned int filter_size = 63;
    const unsigned int image_size = width * height * sizeof(float);

    // Launch configuration
    size_t sharedMemSize = SMEM_SIZE * sizeof(float);
    dim3 blockDim(BLOCK_SIZE / ELEMENTS_PER_THREAD, 1, 1);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, height, 1);

    // Allocate host memory
    float *h_input = (float*)malloc(image_size);
    float *h_output = (float*)malloc(image_size);
    float *h_output_gpu = (float*)malloc(image_size);
    float *h_output_cpu = (float*)malloc(image_size);
    float *h_filter = (float*)malloc(filter_size * sizeof(float));

    // Initialize input image and kernel (example initialization)
    initMatrix(h_input, height, width);
    initMatrix(h_filter, filter_size, 1);

    // Allocate device memory
    float *d_input, *d_output;
    check_cuda_error(hipMalloc(&d_input, image_size), "hipMalloc d_input");
    check_cuda_error(hipMalloc(&d_output, image_size), "hipMalloc d_output");

    // Copy input data to device
    check_cuda_error(hipMemcpy(d_input, h_input, image_size, hipMemcpyHostToDevice), "hipMemcpy H2D input");
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(filter_cm), h_filter, filter_size * sizeof(float)), "hipMemcpyToSymbol kernel");

    // Check computation
    conv_1dhz_cpu(h_input, h_output_cpu, width, height, h_filter, filter_size);
    conv_1d_tile_all<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
    check_cuda_error(hipGetLastError(), "Kernel launch");
    check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    check_cuda_error(hipMemcpy(h_output_gpu, d_output, image_size, hipMemcpyDeviceToHost), "hipMemcpy D2H output");
    compare_results(h_output_cpu, h_output_gpu, width, height, 1e-5f);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
         conv_1d_tile_all<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
        // Wait for kernel to finish
        check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    }
    

    // Create events
    hipEvent_t start, stop;
    check_cuda_error(hipEventCreate(&start), "create event start");
    check_cuda_error(hipEventCreate(&stop), "create event stop");
    // Benchmark
    float elapsed_time;
    float repeats = 100.0f;
    long long flops = 2LL * width * height * filter_size;
    check_cuda_error(hipEventRecord(start), "start event recording");
    for (int i = 0; i < repeats; i++) {
         conv_1d_tile_all<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
    }
    check_cuda_error(hipEventRecord(stop), "stop event recording");
    check_cuda_error(hipEventSynchronize(start), "hipDeviceSynchronize");
    check_cuda_error(hipEventSynchronize(stop), "hipDeviceSynchronize");
    check_cuda_error(hipEventElapsedTime(&elapsed_time, start, stop), "elapsed time");
    printf(
        "Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-6) / elapsed_time
    );
    

    // Free memory
    check_cuda_error(hipFree(d_input), "hipFree d_input");
    check_cuda_error(hipFree(d_output), "hipFree d_output");
    check_cuda_error(hipEventDestroy(start), "hipEventDestroy start");
    check_cuda_error(hipEventDestroy(stop), "hipEventDestroy stop");
    free(h_input);
    free(h_output);
    free(h_filter);
  

    return 0;
}