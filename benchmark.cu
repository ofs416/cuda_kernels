#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

extern "C" { 
#include "cpu_functions.h"
}
#include "gpu_functions.cuh"

#define N 4096  
#define K 4096  
#define M 4096  
#define BLOCK_SIZE 32

int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory (for cpu benchmarks)
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    // Initialize matrices
    srand(time(NULL));
    initMatrix(h_A, M, K);
    initMatrix(h_B, K, N);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsed_time;
    float repeats = 50.0f;
    long long flops = 2LL * M * N * K;

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockDim1D(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridDim4((N + 64 - 1) / 64, (M + 64 - 1) / 64);
    dim3 blockDim4((64 * 64) / 8);
    dim3 gridDim5((N + 128 - 1) / 128, (M + 128 - 1) / 128);
    dim3 blockDim5((128 * 128) / (8 * 8));

    float alpha = 1.0f;
    float beta = 0.0f;

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        gemm<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_gmc<<<gridDim, blockDim1D>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_smem<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_1DBlockTiling<64, 64, 8, 8><<<gridDim4, blockDim4>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_2DBlockTiling<128, 128, 8, 8, 8><<<gridDim5, blockDim5>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, HIP_R_32F,
               N, d_A, HIP_R_32F, K, &beta, d_C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        hipDeviceSynchronize();
        gemm_vectorised<128, 128, 8, 8, 8><<<gridDim5, blockDim5>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    // printf("Benchmarking matrixMultiplicationCPU\n");
    // double cpuTotalTime = 0.0;
    // for (int i = 0; i < 20; i++) {
    //    double startTime = getTime();
    //    matrixMultiplicationCPU(h_A, h_B, h_C_cpu, N, K, M);
    //    double endTime = getTime();
    //    cpuTotalTime += endTime - startTime;
    //}
    // double cpuAvgTime = cpuTotalTime / 20.0;

    // Benchmark CuBlas
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, HIP_R_32F,
               N, d_A, HIP_R_32F, K, &beta, d_C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    double CuBlas_GFLOP = (repeats * flops * 1e-9) / elapsed_time;
    printf(
        "(CuBlas) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        CuBlas_GFLOP, 
        100 * CuBlas_GFLOP / CuBlas_GFLOP);

    // Benchmark GPU implementation 1
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(1) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 2
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_gmc<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(2) Avg time: %f ms, performance: %f GFLOP, %f%% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 2
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_smem<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(3) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 4
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_1DBlockTiling<64, 64, 8, 8><<<gridDim4, blockDim4>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(4) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 5
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_2DBlockTiling<128, 128, 8, 8, 8><<<gridDim5, blockDim5>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(5) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 6
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_vectorised<128, 128, 8, 8, 8><<<gridDim5, blockDim5>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(6) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);


    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
