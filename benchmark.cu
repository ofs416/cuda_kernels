#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

extern "C" { 
#include "cpu_functions.h"
}
#include "gpu_functions.cuh"

#define N 4096  
#define K 4096  
#define M 4096  
#define BLOCK_SIZE 32

#define BM_1D 64
#define BN_1D 64
#define BK_1D 8
#define TM_1D 8

#define BM 128
#define BN 128
#define BK 8
#define TM 8
#define TN 8

int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory (for cpu benchmarks)
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    // Initialize matrices
    srand(time(NULL));
    initMatrix(h_A, M, K);
    initMatrix(h_B, K, N);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsed_time;
    float repeats = 50.0f;
    long long flops = 2LL * M * N * K;

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockDim1D(BLOCK_SIZE * BLOCK_SIZE);

    // Grid and block dimensions for 1D block tiling
    dim3 blockDim1DTiling((BM_1D * BN_1D) / TM_1D); 
    dim3 gridDim1DTiling((N + BN_1D - 1) / BN_1D, (M + BM_1D - 1) / BM_1D);

    // Grid and block dimensions for 2D block tiling and vectorized
    dim3 blockDim2DTiling((BM * BN) / (TM * TN));  // Each thread handles TM x TN elements
    dim3 gridDim2DTiling((N + BN - 1) / BN, (M + BM - 1) / BM);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, HIP_R_32F,
        N, d_A, HIP_R_32F, K, &beta, d_C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        hipDeviceSynchronize();
        gemm<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_gmc<<<gridDim, blockDim1D>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_smem<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_1DBlockTiling<<<gridDim1DTiling, blockDim1DTiling>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_2DBlockTiling<<<gridDim2DTiling, blockDim2DTiling>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        gemm_vectorised<<<gridDim2DTiling, blockDim2DTiling>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    // printf("Benchmarking matrixMultiplicationCPU\n");
    // double cpuTotalTime = 0.0;
    // for (int i = 0; i < 20; i++) {
    //    double startTime = getTime();
    //    matrixMultiplicationCPU(h_A, h_B, h_C_cpu, N, K, M);
    //    double endTime = getTime();
    //    cpuTotalTime += endTime - startTime;
    //}
    // double cpuAvgTime = cpuTotalTime / 20.0;

    // Benchmark CuBlas
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, HIP_R_32F,
               N, d_A, HIP_R_32F, K, &beta, d_C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    double CuBlas_GFLOP = (repeats * flops * 1e-9) / elapsed_time;
    printf(
        "(CuBlas) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        CuBlas_GFLOP, 
        100 * CuBlas_GFLOP / CuBlas_GFLOP);

    // Benchmark GPU implementation 1
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(1) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 2
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_gmc<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(2) Avg time: %f ms, performance: %f GFLOP, %f%% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 2
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_smem<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(3) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 4
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_1DBlockTiling<<<gridDim1DTiling, blockDim1DTiling>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(4) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 5
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_2DBlockTiling<<<gridDim2DTiling, blockDim2DTiling>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(5) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);

    // Benchmark implementation 6
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        gemm_vectorised<<<gridDim2DTiling, blockDim2DTiling>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(6) Avg time: %f ms, performance: %f GFLOP, %f %% \n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time,
        (100 * (repeats * flops * 1e-9) / elapsed_time) / CuBlas_GFLOP);


    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
