#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

extern "C" { 
#include "cpu_functions.h"
}
#include "gpu_functions.h"

#define N 1024  // Number of rows in A and C
#define K 512   // Number of columns in A and rows in B
#define M 1024  // Number of columns in B and C
#define BLOCK_SIZE 32

int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    // Initialize matrices
    srand(time(NULL));
    initMatrix(h_A, M, K);
    initMatrix(h_B, K, N);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim1D(BLOCK_SIZE * BLOCK_SIZE);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);


    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        matrixMultiplicationGPU<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        gemm_gmc<<<gridDim, blockDim1D>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking matrixMultiplicationGPU\n");
    double cpuTotalTime = 0.0;
    for (int i = 0; i < 20; i++) {
        double startTime = getTime();
        matrixMultiplicationGPU<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        double endTime = getTime();
        cpuTotalTime += endTime - startTime;
    }
    double cpuAvgTime = cpuTotalTime / 20.0;

    // Benchmark GPU implementation
    printf("Benchmarking gemm_gmc\n");
    double gpuTotalTime = 0.0;
    for (int i = 0; i < 20; i++) {
        double startTime = getTime();
        gemm_gmc<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        double endTime = getTime();
        gpuTotalTime += endTime - startTime;
    }
    double gpuAvgTime = gpuTotalTime / 20.0;

    // Print results
    printf("matrixMultiplicationGPU average time: %f microseconds\n", (cpuAvgTime * 1e6f));
    printf("gemm_gmc average time: %f microseconds\n", (gpuAvgTime * 1e6f));
    printf("Speedup: %fx\n", cpuAvgTime / gpuAvgTime);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}