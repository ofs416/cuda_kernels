#include "hip/hip_runtime.h"
// TODO: NEED TO RESEARCH AND FIX THIS

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

extern "C" { 
#include "cpu_functions.h"
}

#define BLOCK_SIZE 1024
#define MAX_FILTER_SIZE 63
#define ELEMENTS_PER_THREAD 2
#define ACTUAL_BLOCK_SIZE (BLOCK_SIZE/ELEMENTS_PER_THREAD) // 512 threads per block
#define SMEM_SIZE (BLOCK_SIZE + MAX_FILTER_SIZE - 1)

__constant__ float filter_cm[MAX_FILTER_SIZE];

__global__ void conv_1d_vectorised(float *input, float *output, int width, 
                                 int height, int filter_size, bool transpose) {
    extern __shared__ float shared_mem[];
    
    const int tid = threadIdx.x;
    const int row = blockIdx.y;
    const int base_col = blockIdx.x * BLOCK_SIZE + tid * ELEMENTS_PER_THREAD;
    const int radius = filter_size / 2;
    
    int trans_width = transpose ? height : width;
    int trans_height = transpose ? width : height;
    int input_row = transpose ? base_col : row;
    
    if (input_row >= trans_height) return;

    // Load data into shared memory
    const int block_start = blockIdx.x * BLOCK_SIZE - radius;
    const int elements_to_load = BLOCK_SIZE + filter_size - 1;
    const int vector_elements = (elements_to_load + 4 - 1) / 4;
    const int loads_per_thread = (vector_elements + ACTUAL_BLOCK_SIZE - 1) / ACTUAL_BLOCK_SIZE;

    // Base pointer for vectorized loads
    const float4* input4 = reinterpret_cast<const float4*>(input + input_row * trans_width);

    // Load phase - populate shared memory
    #pragma unroll
    for (int i = 0; i < loads_per_thread; ++i) {
        const int vec_offset = tid + (i * ACTUAL_BLOCK_SIZE);
        if (vec_offset < vector_elements) {
            const int global_vec_idx = (block_start + vec_offset * 4) / 4;
            const int global_pos = global_vec_idx * 4;
            
            float4 reg_vec = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            
            if (global_pos >= 0 && global_pos + 3 < trans_width) {
                reg_vec = input4[global_vec_idx];
            } else {
                // Handle boundary conditions manually
                for (int j = 0; j < 4; ++j) {
                    const int elem_idx = global_pos + j;
                    if (elem_idx >= 0 && elem_idx < trans_width) {
                        reinterpret_cast<float*>(&reg_vec)[j] = 
                            input[input_row * trans_width + elem_idx];
                    }
                }
            }
            
            // Store to shared memory
            shared_mem[vec_offset * 4 + 0] = reg_vec.x;
            shared_mem[vec_offset * 4 + 1] = reg_vec.y;
            shared_mem[vec_offset * 4 + 2] = reg_vec.z;
            shared_mem[vec_offset * 4 + 3] = reg_vec.w;
        }
    }
    __syncthreads();
    // Compute convolution for each element handled by this thread
    float results[ELEMENTS_PER_THREAD];
    #pragma unroll
    for (int e = 0; e < ELEMENTS_PER_THREAD; e++) {
        const int shared_idx = tid * ELEMENTS_PER_THREAD + e;
        float sum = 0.0f;
        
        #pragma unroll
        for (int f = 0; f < filter_size; f++) {
            sum += shared_mem[shared_idx + f] * filter_cm[f];
        }
        
        results[e] = sum;
    }

    // Write results
    // Each thread processes ELEMENTS_PER_THREAD consecutive elements
    #pragma unroll
    for (int offset = 0; offset < ELEMENTS_PER_THREAD; offset++) {
        int input_col = base_col + offset;
        if (input_col < trans_width) {
            // Store result
            if (transpose) {
                output[input_col * trans_height + input_row] = results[offset];
            } else {
                output[input_row * trans_width + input_col] = results[offset];
            }
        }
    }
}


void check_cuda_error(hipError_t error, const char *function_name) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function_name, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Image and kernel parameters
    const unsigned int width = 4096;
    const unsigned int height = 4096;
    const unsigned int filter_size = 63;
    const unsigned int image_size = width * height * sizeof(float);

    // Launch configuration
    size_t sharedMemSize = SMEM_SIZE * sizeof(float);
    dim3 blockDim(BLOCK_SIZE / ELEMENTS_PER_THREAD, 1, 1);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, height, 1);

    // Allocate host memory
    float *h_input = (float*)malloc(image_size);
    float *h_output = (float*)malloc(image_size);
    float *h_output_gpu = (float*)malloc(image_size);
    float *h_output_cpu = (float*)malloc(image_size);
    float *h_filter = (float*)malloc(filter_size * sizeof(float));

    // Initialize input image and kernel (example initialization)
    initMatrix(h_input, height, width);
    initMatrix(h_filter, filter_size, 1);

    // Allocate device memory
    float *d_input, *d_output;
    check_cuda_error(hipMalloc(&d_input, image_size), "hipMalloc d_input");
    check_cuda_error(hipMalloc(&d_output, image_size), "hipMalloc d_output");

    // Copy input data to device
    check_cuda_error(hipMemcpy(d_input, h_input, image_size, hipMemcpyHostToDevice), "hipMemcpy H2D input");
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(filter_cm), h_filter, filter_size * sizeof(float)), "hipMemcpyToSymbol kernel");

    // Check computation
    conv_1dhz_cpu(h_input, h_output_cpu, width, height, h_filter, filter_size);
    conv_1d_vectorised<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
    check_cuda_error(hipGetLastError(), "Kernel launch");
    check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    check_cuda_error(hipMemcpy(h_output_gpu, d_output, image_size, hipMemcpyDeviceToHost), "hipMemcpy D2H output");
    compare_results(h_output_cpu, h_output_gpu, width, height, 1e-3f);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
         conv_1d_vectorised<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
        // Wait for kernel to finish
        check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    }
    

    // Create events
    hipEvent_t start, stop;
    check_cuda_error(hipEventCreate(&start), "create event start");
    check_cuda_error(hipEventCreate(&stop), "create event stop");
    // Benchmark
    float elapsed_time;
    float repeats = 100.0f;
    long long flops = 2LL * width * height * filter_size;
    check_cuda_error(hipEventRecord(start), "start event recording");
    for (int i = 0; i < repeats; i++) {
         conv_1d_vectorised<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
    }
    check_cuda_error(hipEventRecord(stop), "stop event recording");
    check_cuda_error(hipEventSynchronize(start), "hipDeviceSynchronize");
    check_cuda_error(hipEventSynchronize(stop), "hipDeviceSynchronize");
    check_cuda_error(hipEventElapsedTime(&elapsed_time, start, stop), "elapsed time");
    printf(
        "Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-6) / elapsed_time
    );
    

    // Free memory
    check_cuda_error(hipFree(d_input), "hipFree d_input");
    check_cuda_error(hipFree(d_output), "hipFree d_output");
    check_cuda_error(hipEventDestroy(start), "hipEventDestroy start");
    check_cuda_error(hipEventDestroy(stop), "hipEventDestroy stop");
    free(h_input);
    free(h_output);
    free(h_filter);
  

    return 0;
}