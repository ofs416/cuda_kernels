#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

extern "C" { 
#include "cpu_functions.h"
}

#define BLOCK_SIZE 16
#define MAX_FILTER_SIZE 63
#define SMEM_SIZE (BLOCK_SIZE + MAX_FILTER_SIZE - 1)

__constant__ float filter_cm[MAX_FILTER_SIZE];

__global__ void conv_1dhz_smem(float *input, float *output, int width, 
                              int height, int filter_size) {
    extern __shared__ float shared_mem[];
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int row = blockIdx.y * BLOCK_SIZE + ty;
    const int col = blockIdx.x * BLOCK_SIZE + tx;
    const int radius = filter_size / 2;
    
    if (row >= height) return;

    float* row_shared = &shared_mem[ty * SMEM_SIZE];
    const int block_start = blockIdx.x * BLOCK_SIZE - radius;
    for (int i = tx; i < (BLOCK_SIZE + filter_size - 1); i += BLOCK_SIZE) {
        int global_idx = block_start + i;
        if (global_idx >= 0 && global_idx < width) {
            row_shared[i] = input[row * width + global_idx];
        } else {
            row_shared[i] = 0.0f;
        }
    }
    __syncthreads();
    
    // Compute convolution only for valid output positions
    if (col < width) {
        float sum = 0.0f;
        #pragma unroll
        for (int i = 0; i < filter_size; i++) {
            sum += row_shared[tx + i] * filter_cm[i];
        }
        output[row * width + col] = sum;
    }
}


void check_cuda_error(hipError_t error, const char *function_name) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function_name, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Image and kernel parameters
    const unsigned int width = 4096;
    const unsigned int height = 4096;
    const unsigned int filter_size = 63;
    const unsigned int image_size = width * height * sizeof(float);

    // Allocate host memory
    float *h_input = (float*)malloc(image_size);
    float *h_output = (float*)malloc(image_size);
    float *h_output_gpu = (float*)malloc(image_size);
    float *h_output_cpu = (float*)malloc(image_size);
    float *h_filter = (float*)malloc(filter_size * sizeof(float));

    // Initialize input image and kernel (example initialization)
    initMatrix(h_input, height, width);
    initMatrix(h_filter, filter_size, 1);

    // Allocate device memory
    float *d_input, *d_output;
    check_cuda_error(hipMalloc(&d_input, image_size), "hipMalloc d_input");
    check_cuda_error(hipMalloc(&d_output, image_size), "hipMalloc d_output");

    // Copy input data to device
    check_cuda_error(hipMemcpy(d_input, h_input, image_size, hipMemcpyHostToDevice), "hipMemcpy H2D input");
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(filter_cm), h_filter, filter_size * sizeof(float)), "hipMemcpyToSymbol kernel");

    // Launch configuration
    size_t shared_mem_size = BLOCK_SIZE * SMEM_SIZE * sizeof(float);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, 
            (height + blockDim.y - 1) / blockDim.y);


    // Check computation
    // Compute convolution on CPU
    conv_1dhz_cpu(h_input, h_output_cpu, width, height, h_filter, filter_size);
    // compute convolution with custom kernel
    conv_1dhz_smem<<<gridDim, blockDim, shared_mem_size>>>(d_input, d_output, width, height, filter_size);
    // Check for kernel launch errors
    check_cuda_error(hipGetLastError(), "Kernel launch");
    // Wait for kernel to finish
    check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    // Copy result back to host
    check_cuda_error(hipMemcpy(h_output_gpu, d_output, image_size, hipMemcpyDeviceToHost), "hipMemcpy D2H output");
    // Compare CPU and GPU results
    compare_results(h_output_cpu, h_output_gpu, width, height, 1e-5f);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        conv_1dhz_smem<<<gridDim, blockDim, shared_mem_size>>>(d_input, d_output, width, height, filter_size);
        // Wait for kernel to finish
        check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    }
    

    // Create events
    hipEvent_t start, stop;
    check_cuda_error(hipEventCreate(&start), "create event start");
    check_cuda_error(hipEventCreate(&stop), "create event stop");
    // Benchmark
    float elapsed_time;
    float repeats = 100.0f;
    long long flops = 2LL * width * height * filter_size;
    check_cuda_error(hipEventRecord(start), "start event recording");
    for (int i = 0; i < repeats; i++) {
        conv_1dhz_smem<<<gridDim, blockDim, shared_mem_size>>>(d_input, d_output, width, height, filter_size);
    }
    check_cuda_error(hipEventRecord(stop), "stop event recording");
    check_cuda_error(hipEventSynchronize(start), "hipDeviceSynchronize");
    check_cuda_error(hipEventSynchronize(stop), "hipDeviceSynchronize");
    check_cuda_error(hipEventElapsedTime(&elapsed_time, start, stop), "elapsed time");
    printf(
        "Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time
    );
    

    // Free memory
    check_cuda_error(hipFree(d_input), "hipFree d_input");
    check_cuda_error(hipFree(d_output), "hipFree d_output");
    check_cuda_error(hipEventDestroy(start), "hipEventDestroy start");
    check_cuda_error(hipEventDestroy(stop), "hipEventDestroy stop");
    free(h_input);
    free(h_output);
    free(h_filter);
  

    return 0;
}