#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

extern "C" { 
#include "cpu_functions.h"
}
#include "cnn_kernels.cuh"

#define N 1024  
#define K 5  
#define M 1024 
#define BLOCK_SIZE 32

int main() {
    float *h_A, *h_B;
    float *d_A, *d_B, *d_C;
    int size_A = M * N * sizeof(float);
    int size_B = K * K * sizeof(float);
    int size_C = (M + 1 - K) * (N + 1 - K) * sizeof(float);

    // Allocate host memory (for cpu benchmarks)
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);

    // Initialize matrices
    initMatrix(h_A, M, K);
    initMatrix(h_B, K, K);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsed_time;
    float repeats = 50.0f;
    long long flops = 2LL * (M + 1 - K) * (N + 1 - K) * K * K;

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((M - K) / BLOCK_SIZE, (M - K) / BLOCK_SIZE);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        conv_naive<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
    }

    // Implementation 1
    hipEventRecord(start);
    for (int i = 0; i < repeats; i++) {
        conv_naive<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(
        "(1) Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-9) / elapsed_time
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
}