#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

extern "C" { 
#include "cpu_functions.h"
}

#define BLOCK_SIZE 1024
#define MAX_FILTER_SIZE 63
#define ELEMENTS_PER_THREAD 2
#define ACTUAL_BLOCK_SIZE (BLOCK_SIZE/ELEMENTS_PER_THREAD) // Now 64 threads per block
#define SMEM_SIZE (BLOCK_SIZE + MAX_FILTER_SIZE - 1)

__constant__ float filter_cm[MAX_FILTER_SIZE];

__global__ void conv_1d_vectorised(float *input, float *output, int width, 
                                        int height, int filter_size, bool transpose) {
    
}



void check_cuda_error(hipError_t error, const char *function_name) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function_name, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Image and kernel parameters
    const unsigned int width = 4096;
    const unsigned int height = 4096;
    const unsigned int filter_size = 63;
    const unsigned int image_size = width * height * sizeof(float);

    // Launch configuration
    size_t sharedMemSize = SMEM_SIZE * sizeof(float);
    dim3 blockDim(BLOCK_SIZE / ELEMENTS_PER_THREAD, 1, 1);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, height, 1);

    // Allocate host memory
    float *h_input = (float*)malloc(image_size);
    float *h_output = (float*)malloc(image_size);
    float *h_output_gpu = (float*)malloc(image_size);
    float *h_output_cpu = (float*)malloc(image_size);
    float *h_filter = (float*)malloc(filter_size * sizeof(float));

    // Initialize input image and kernel (example initialization)
    initMatrix(h_input, height, width);
    initMatrix(h_filter, filter_size, 1);

    // Allocate device memory
    float *d_input, *d_output;
    check_cuda_error(hipMalloc(&d_input, image_size), "hipMalloc d_input");
    check_cuda_error(hipMalloc(&d_output, image_size), "hipMalloc d_output");

    // Copy input data to device
    check_cuda_error(hipMemcpy(d_input, h_input, image_size, hipMemcpyHostToDevice), "hipMemcpy H2D input");
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(filter_cm), h_filter, filter_size * sizeof(float)), "hipMemcpyToSymbol kernel");

    // Check computation
    conv_1dhz_cpu(h_input, h_output_cpu, width, height, h_filter, filter_size);
    conv_1d_vectorised<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
    check_cuda_error(hipGetLastError(), "Kernel launch");
    check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    check_cuda_error(hipMemcpy(h_output_gpu, d_output, image_size, hipMemcpyDeviceToHost), "hipMemcpy D2H output");
    compare_results(h_output_cpu, h_output_gpu, width, height, 1e-5f);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
         conv_1d_vectorised<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
        // Wait for kernel to finish
        check_cuda_error(hipDeviceSynchronize(), "hipDeviceSynchronize");
    }
    

    // Create events
    hipEvent_t start, stop;
    check_cuda_error(hipEventCreate(&start), "create event start");
    check_cuda_error(hipEventCreate(&stop), "create event stop");
    // Benchmark
    float elapsed_time;
    float repeats = 100.0f;
    long long flops = 2LL * width * height * filter_size;
    check_cuda_error(hipEventRecord(start), "start event recording");
    for (int i = 0; i < repeats; i++) {
         conv_1d_vectorised<<<gridDim, blockDim, sharedMemSize >>>(d_input, d_output, width, height, filter_size, false);
    }
    check_cuda_error(hipEventRecord(stop), "stop event recording");
    check_cuda_error(hipEventSynchronize(start), "hipDeviceSynchronize");
    check_cuda_error(hipEventSynchronize(stop), "hipDeviceSynchronize");
    check_cuda_error(hipEventElapsedTime(&elapsed_time, start, stop), "elapsed time");
    printf(
        "Avg time: %f ms, performance: %f GFLOP\n", 
        elapsed_time / repeats, 
        (repeats * flops * 1e-6) / elapsed_time
    );
    

    // Free memory
    check_cuda_error(hipFree(d_input), "hipFree d_input");
    check_cuda_error(hipFree(d_output), "hipFree d_output");
    check_cuda_error(hipEventDestroy(start), "hipEventDestroy start");
    check_cuda_error(hipEventDestroy(stop), "hipEventDestroy stop");
    free(h_input);
    free(h_output);
    free(h_filter);
  

    return 0;
}