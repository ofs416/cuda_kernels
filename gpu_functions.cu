#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "cpu_functions.h"

#define N 256  // Number of rows in A and C
#define K 512   // Number of columns in A and rows in B
#define M 256  // Number of columns in B and C
#define BLOCK_SIZE 32

// CUDA kernel for matrix addition
__global__ void matrixAdditionGPU (float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        C[n * row + col] = A[n * row + col] + B[n * row + col];
    }
}

// CUDA kernel for matrix multiplication (N x K) @ (K x M)
__global__ void matrixMultiplicationGPU (float *A, float *B, float *C, int n, int k, int m) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < m) {
        float sum = 0.0f;
        for (int l = 0; l < k; l++) {
            sum += A[row * k + l] * B[l * m + col];
        }
        C[m * row + col] = sum;
    }
}

int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    // Initialize matrices
    srand(time(NULL));
    initMatrix(h_A, M, K);
    initMatrix(h_B, K, N);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        matrixMultiplicationCPU(h_A, h_B, h_C_cpu, N, K, M);
        matrixMultiplicationGPU<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU\n");
    double cpuTotalTime = 0.0;
    for (int i = 0; i < 20; i++) {
        double startTime = getTime();
        matrixMultiplicationCPU(h_A, h_B, h_C_cpu, N, K, M);
        double endTime = getTime();
        cpuTotalTime += endTime - startTime;
    }
    double cpuAvgTime = cpuTotalTime / 20.0;

    // Benchmark GPU implementation
    printf("Benchmarking GPU\n");
    double gpuTotalTime = 0.0;
    for (int i = 0; i < 20; i++) {
        double startTime = getTime();
        matrixMultiplicationGPU<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, K, M);
        hipDeviceSynchronize();
        double endTime = getTime();
        gpuTotalTime += endTime - startTime;
    }
    double gpuAvgTime = gpuTotalTime / 20.0;

    // Print results
    printf("CPU average time: %f microseconds\n", (cpuAvgTime * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpuAvgTime * 1e6f));
    printf("Speedup: %fx\n", cpuAvgTime / gpuAvgTime);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
