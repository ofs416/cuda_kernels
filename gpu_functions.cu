#include "hip/hip_runtime.h"
#include "gpu_functions.h"

#define BLOCK_SIZE 32

// CUDA kernel for matrix addition
__global__ void matrixAdditionGPU (float *A, float *B, float *C, int n) {
    const uint row = blockIdx.y * blockDim.y + threadIdx.y;
    const uint col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        C[n * row + col] = A[n * row + col] + B[n * row + col];
    }
}

// Naive CUDA kernel for matrix multiplication (N x K) @ (K x M)
__global__ void matrixMultiplicationGPU (float *A, float *B, float *C, int n, int k, int m) {
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int l = 0; l < k; l++) {
            sum += A[row * k + l] * B[l * n + col];
        }
        C[n * row + col] = sum;
    }
}

// Global Memory Coalescing CUDA kernel for matrix multiplication (N x K) @ (K x M)
// Increases performance by grouping memory accesses of threads that are in the same warp
// and executed as one
// Each warp contains 32 threads and memory accesses can be 32B, 64B and 128B
// To take advantage of 128B single access, the floats should be conseecutive in memory
// and aligned in access (but the accesses don’t have to be consecutive within-warp)
__global__ void gemm_gmc (float *A, float *B, float *C, int n, int k, int m) {
    const uint row = blockIdx.x * BLOCK_SIZE + (threadIdx.x / BLOCK_SIZE);
    const uint col = blockIdx.y * BLOCK_SIZE + (threadIdx.x % BLOCK_SIZE);

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int l = 0; l < k; l++) {
            sum += A[row * k + l] * B[l * n + col];
        }
        C[n * row + col] = sum;
    }
}
